#include "hip/hip_runtime.h"
#include <stdint.h>


#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void calcHistogram(uint8_t* srcImage, double * resultHistogram, size_t size, size_t chunk_cnt){

    int globalIdx = 0;
    globalIdx  = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    globalIdx += (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y*blockDim.z;

    double localHistogram[255];
    for (size_t i = 0; i < 255; ++i){
        localHistogram[i] = 0.;
    }

    size_t chunkSize = size / chunk_cnt;
    size_t startIdx = chunkSize * globalIdx;
    size_t endtIdx = startIdx + chunkSize;
    if (endtIdx > size){
        endtIdx = size;
    } 
    //printf(" %d \t %d \t %d\n",(int)globalIdx, (int)startIdx, (int)endtIdx);
    for(size_t i = startIdx; i < endtIdx; ++i){
        localHistogram[srcImage[i]] += 1.;
    }

    for(size_t i = 0; i < 255; ++i){
        atomicAdd(&resultHistogram[i], localHistogram[i]);

    }
}

int main() {
    int width, height, bpp;

    uint8_t* h_grayImage = stbi_load("../data/lena_gray_noise.png", &width, &height, &bpp, 1);
    size_t imageSize = height * width * sizeof(uint8_t);



    double * h_histogram= (double *) malloc(255 * sizeof(double));
    for (size_t i =0; i< 255; ++i){
        h_histogram[i] = 0.;

    }


 
    uint8_t* d_grayImage;
    double * d_histogram;
 
    hipMalloc(&d_grayImage, imageSize);
    hipMalloc(&d_histogram, 255 * sizeof(double));
 

 
    hipMemcpy(d_grayImage, h_grayImage, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, h_histogram, 255 * sizeof(double), hipMemcpyHostToDevice);

    calcHistogram<<<1, 16>>>(d_grayImage, d_histogram, imageSize, 16);      
    hipDeviceSynchronize();    
    
    hipMemcpy(h_histogram, d_histogram, 255 * sizeof(double), hipMemcpyDeviceToHost);
    

    stbi_image_free(h_grayImage);


    double histSumm = 0.;
    for (size_t i =0; i< 255; ++i){
        histSumm += h_histogram[i];

    }

    printf(" color \t normalized value\n");

    for (size_t i =0; i< 255; ++i){
        h_histogram[i] /= histSumm;
        printf("color %d \t %f\n", (int)i, h_histogram[i]);

    }

    free(h_histogram); 

 
    hipFree(d_grayImage);
    hipFree(d_histogram);  
 

    return 0;
}