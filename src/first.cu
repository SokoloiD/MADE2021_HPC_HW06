#include "hip/hip_runtime.h"
#include <stdint.h>


#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void matrixFilter(uint8_t* srcImage, uint8_t* dstImage, int w, int h, size_t size, int dw, int  dh, uint8_t * filter){

    int globalIdx = 0, x,y, summ;
    globalIdx  = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    globalIdx += (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y*blockDim.z;
    int hIdx =  globalIdx / (w / dw);
    int wIdx =  globalIdx % (w / dw);
    for(int i=0; i< dw; ++i){
        x = dw * wIdx + i;
        if (x > w){
            continue;
        }
        for(int j =0; j < dh; j++){
            y = dh* hIdx + j;
            if (y > h){
                continue;
            }
            summ = 0;

            for(int k=-2;k<=2; k++){
                for(int l=-2; l<=2; l++){

                    summ += srcImage[ x + k +  (y + l) * w] * filter[2 + k + (2 +l) *5];                
                }
            dstImage[x + y * w] = summ /25;
            }
        }
    }

}

int main() {
    int width, height, bpp;

    uint8_t* h_grayImage = stbi_load("../data/lena_gray.png", &width, &height, &bpp, 1);
    size_t imageSize = height * width * sizeof(uint8_t);



    uint8_t * h_newImage = (uint8_t *) malloc(imageSize);
    uint8_t * h_filter = (uint8_t *) malloc(25 * sizeof(uint8_t));
    uint8_t* d_grayImage;
    uint8_t* d_newImage;
    uint8_t* d_filter;
    hipMalloc(&d_grayImage, imageSize);
    hipMalloc(&d_newImage, imageSize);
    hipMalloc(&d_filter, 25 * sizeof(uint8_t));


    int gridCnt = 4;
    int dw = width / gridCnt;
    int dh = height/ gridCnt;

    // Фильтр усиления контуров
    for( int i = 0; i< 25; ++i){
        h_filter[i]=1;
    }
    h_filter[12]=0;

    hipMemcpy(d_filter, h_filter, 25, hipMemcpyHostToDevice);
    hipMemcpy(d_grayImage, h_grayImage, imageSize, hipMemcpyHostToDevice);

    matrixFilter<<<1, 16>>>(d_grayImage, d_newImage, width, height, imageSize, dw, dh, d_filter );      
    hipDeviceSynchronize();

    hipMemcpy(h_newImage, d_newImage, imageSize, hipMemcpyDeviceToHost);
    


    stbi_write_png("../data/lena_123.png", width, height, 1, h_newImage, width*1);

    stbi_image_free(h_grayImage);
    free(h_newImage);
    free(h_filter);

 
    hipFree(d_filter);
    hipFree(d_grayImage);
    hipFree(d_newImage);

    return 0;
}