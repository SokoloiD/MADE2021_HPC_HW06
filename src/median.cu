#include "hip/hip_runtime.h"
#include <stdint.h>


#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void medianFilter(uint8_t* srcImage, uint8_t* dstImage, int w, int h, size_t size, int dw, int  dh){

    int globalIdx = 0, x,y;
    globalIdx  = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    globalIdx += (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y*blockDim.z;
    int hIdx =  globalIdx / (w / dw);
    int wIdx =  globalIdx % (w / dw);
    uint8_t filterBuffer[25];
    for(int i=0; i< dw; ++i){
        x = dw * wIdx + i;
        if (x > w){
            continue;
        }
        for(int j =0; j < dh; j++){
            y = dh* hIdx + j;
            if (y > h){
                continue;
            }
 
            // заполняем фильтр
            for(int k=-2;k<=2; k++){
                for(int l=-2; l<=2; l++){

                    filterBuffer[2 + k + (2 +l) *5] = srcImage[ x + k +  (y + l) * w]; 
        
                }
            // абы как (пузырьком) сортируем
            uint8_t tmp;        
            for(int k = 0; k < 25; ++k){
                for(int l = 0; l < 25 - k; ++l){
                    if (filterBuffer[l] < filterBuffer[l + 1]){
                        tmp = filterBuffer[l];
                        filterBuffer[l] =  filterBuffer[l + 1];
                        filterBuffer[l + 1] = tmp;
                    }

                }
            }




            dstImage[x + y * w] = filterBuffer[12];
            }
        }
    }

}

int main() {
    int width, height, bpp;

    uint8_t* h_grayImage = stbi_load("../data/lena_gray_noise.png", &width, &height, &bpp, 1);
    size_t imageSize = height * width * sizeof(uint8_t);



    uint8_t * h_newImage = (uint8_t *) malloc(imageSize);
 
    uint8_t* d_grayImage;
    uint8_t* d_newImage;
 
    hipMalloc(&d_grayImage, imageSize);
    hipMalloc(&d_newImage, imageSize);
  


    int gridCnt = 4;
    int dw = width / gridCnt;
    int dh = height/ gridCnt;

 

 
    hipMemcpy(d_grayImage, h_grayImage, imageSize, hipMemcpyHostToDevice);
    medianFilter<<<1, 16>>>(d_grayImage, d_newImage, width, height, imageSize, dw, dh);      
    hipMemcpy(h_newImage, d_newImage, imageSize, hipMemcpyDeviceToHost);
    


    stbi_write_png("../data/lena_gray_median.png", width, height, 1, h_newImage, width*1);

    stbi_image_free(h_grayImage);
    free(h_newImage);
 

    hipDeviceSynchronize();
 
    hipFree(d_grayImage);
    hipFree(d_newImage);

    return 0;
}